#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <sys/types.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

__global__ 
void performUpdatesKernel(int *h_Rnew, int *h_R,int *h_Gnew, int *h_G,int *h_Bnew, int *h_B,int colsize, int rowsize)
{
	int row = 0, col = 0;
    for(row=0;row<rowsize;row++){
			for (col=0;col<colsize;col++){	
				if (row != 0 && row != (rowsize-1) && col != 0 && col != (colsize-1)){
					h_Rnew[row][col] = (h_R[row+1][col]+h_R[row-1][col]+h_R[row][col+1]+h_R[row][col-1])/4;
					h_Gnew[row][col] = (h_G[row+1][col]+h_G[row-1][col]+h_G[row][col+1]+h_G[row][col-1])/4;
					h_Bnew[row][col] = (h_B[row+1][col]+h_B[row-1][col]+h_B[row][col+1]+h_B[row][col-1])/4;
				}
				else if (row == 0 && col != 0 && col != (colsize-1)){
					h_Rnew[row][col] = (h_R[row+1][col]+h_R[row][col+1]+h_R[row][col-1])/3;
					h_Gnew[row][col] = (h_G[row+1][col]+h_G[row][col+1]+h_G[row][col-1])/3;
					h_Bnew[row][col] = (h_B[row+1][col]+h_B[row][col+1]+h_B[row][col-1])/3;
				}
				else if (row == (rowsize-1) && col != 0 && col != (colsize-1)){
					h_Rnew[row][col] = (h_R[row-1][col]+h_R[row][col+1]+h_R[row][col-1])/3;
					h_Gnew[row][col] = (h_G[row-1][col]+h_G[row][col+1]+h_G[row][col-1])/3;
					h_Bnew[row][col] = (h_B[row-1][col]+h_B[row][col+1]+h_B[row][col-1])/3;
				}
				else if (col == 0 && row != 0 && row != (rowsize-1)){
					h_Rnew[row][col] = (h_R[row+1][col]+h_R[row-1][col]+h_R[row][col+1])/3;
					h_Gnew[row][col] = (h_G[row+1][col]+h_G[row-1][col]+h_G[row][col+1])/3;
					h_Bnew[row][col] = (h_B[row+1][col]+h_B[row-1][col]+h_B[row][col+1])/3;
				}
				else if (col == (colsize-1) && row != 0 && row != (rowsize-1)){
					h_Rnew[row][col] = (h_R[row+1][col]+h_R[row-1][col]+h_R[row][col-1])/3;
					h_Gnew[row][col] = (h_G[row+1][col]+h_G[row-1][col]+h_G[row][col-1])/3;
					h_Bnew[row][col] = (h_B[row+1][col]+h_B[row-1][col]+h_B[row][col-1])/3;
				}
				else if (row==0 &&col==0){
					h_Rnew[row][col] = (h_R[row][col+1]+h_R[row+1][col])/2;
					h_Gnew[row][col] = (h_G[row][col+1]+h_G[row+1][col])/2;
					h_Bnew[row][col] = (h_B[row][col+1]+h_B[row+1][col])/2;
				}
				else if (row==0 &&col==(colsize-1)){
					h_Rnew[row][col] = (h_R[row][col-1]+h_R[row+1][col])/2;
					h_Gnew[row][col] = (h_G[row][col-1]+h_G[row+1][col])/2;
					h_Bnew[row][col] = (h_B[row][col-1]+h_B[row+1][col])/2;
				}
				else if (row==(rowsize-1) &&col==0){
					h_Rnew[row][col] = (h_R[row][col+1]+h_R[row-1][col])/2;
					h_Gnew[row][col] = (h_G[row][col+1]+h_G[row-1][col])/2;
					h_Bnew[row][col] = (h_B[row][col+1]+h_B[row-1][col])/2;
				}
				else if (row==(rowsize-1) &&col==(colsize-1)){
					h_Rnew[row][col] = (h_R[row][col-1]+h_R[row-1][col])/2;
					h_Gnew[row][col] = (h_G[row][col-1]+h_G[row-1][col])/2;
					h_Bnew[row][col] = (h_B[row][col-1]+h_B[row-1][col])/2;
				}		
			}
		}
}
__global__
void doCopyKernel(int *h_Rnew, int *h_R,int *h_Gnew, int *h_G,int *h_Bnew, int *h_B,int colsize, int rowsize)
{
	int row = 0, col = 0;
    for(row=0;row<rowsize;row++){
			for (col=0;col<colsize;col++){
			    h_R[row][col] = h_Rnew[row][col];
			    h_G[row][col] = h_Gnew[row][col];
			    h_B[row][col] = h_Bnew[row][col];
			}
		}
}


int main (int argc, const char * argv[]) {
	static int const maxlen = 200, rowsize = 521, colsize = 428, linelen = 12;
	char str[maxlen], lines[5][maxlen];
	FILE *fp, *fout;
	int nlines = 0;
	unsigned int h1, h2, h3;
	char *sptr;
	int R[rowsize][colsize], G[rowsize][colsize], B[rowsize][colsize];
	int Rnew[rowsize][colsize], Gnew[rowsize][colsize], Bnew[rowsize][colsize];
	int row = 0, col = 0, nblurs, lineno=0, k;
	struct timeval tim;
	
	fp = fopen("David.ps", "r");
 
	while(! feof(fp))
	{
		fscanf(fp, "\n%[^\n]", str);
		if (nlines < 5) {strcpy((char *)lines[nlines++],(char *)str);}
		else{
			for (sptr=&str[0];*sptr != '\0';sptr+=6){
				sscanf(sptr,"%2x",&h1);
				sscanf(sptr+2,"%2x",&h2);
				sscanf(sptr+4,"%2x",&h3);
				
				if (col==colsize){
					col = 0;
					row++;
				}
				if (row < rowsize) {
					R[row][col] = h1;
					G[row][col] = h2;
					B[row][col] = h3;
				}
				col++;
			}
		}
	}
	fclose(fp);


	fout= fopen("DavidBlur.ps", "w");
	for (k=0;k<nlines;k++) fprintf(fout,"\n%s", lines[k]);
	fprintf(fout,"\n");
	for(row=0;row<rowsize;row++){
		for (col=0;col<colsize;col++){
			fprintf(fout,"%02x%02x%02x",R[row][col],G[row][col],B[row][col]);
			lineno++;
			if (lineno==linelen){
				fprintf(fout,"\n");
				lineno = 0;
			}
		}
	}
	fclose(fout);
	doBlur (Rnew, R, Gnew, G,Bnew, B,colsize,rowsize);
    return 0;
}
	
	int doBlur (int *Rnew, int *R,int *Gnew, int *G,int *Bnew, int *B,int colsize, int rowsize) {
	nblurs = 10;
	printf("%i\n",nblurs);
	gettimeofday(&tim, NULL);

	double t1=tim.tv_sec+(tim.tv_usec/1000000.0);
	int *h_R[rowsize][colsize], *h_G[rowsize][colsize], *h_B[rowsize][colsize];
	int *h_Rnew[rowsize][colsize], *h_Gnew[rowsize][colsize], *h_Bnew[rowsize][colsize];

	h_R = (int **)malloc((sizeof(int*)*(myrowsize)));
	h_G = (int **)malloc((sizeof(int*)*(myrowsize)));
	h_B = (int **)malloc((sizeof(int*)*(myrowsize)));


	// // memset(h_R, 0, sizeof h_R);
	// // memset(h_G, 0, sizeof h_G);
	// // memset(h_B, 0, sizeof h_B);


	h_Rnew = (int **)malloc((sizeof(int*)*(myrowsize)));
	h_Gnew = (int **)malloc((sizeof(int*)*(myrowsize)));
	h_Bnew = (int **)malloc((sizeof(int*)*(myrowsize)));

    
 //    memset(h_R, 0, sizeof h_R);
	// memset(h_G, 0, sizeof h_G);
	// memset(h_B, 0, sizeof h_B);

	int sizef = sizeof(int)*colsize*rowsize;
    hipMalloc((void **)&h_Rnew,sizef);
    hipMalloc((void **)&h_R,sizef);

	hipMalloc((void **)&h_Gnew,sizef);
    hipMalloc((void **)&h_G,sizef);

	hipMalloc((void **)&h_Bnew,sizef);
    hipMalloc((void **)&h_B,sizef);

    hipMemcpy(h_R,R,sizef,hipMemcpyHostToDevice);
    hipMemcpy(h_G,G,sizef,hipMemcpyHostToDevice);
    hipMemcpy(h_B,B,sizef,hipMemcpyHostToDevice);


    for(k=0;k<nblurs;k++){
		performUpdatesKernel(h_Rnew,h_R,h_Gnew,h_G,h_Bnew,h_B,colsize,rowsize);
        doCopyKernel(h_Rnew,h_R,h_Gnew,h_G,h_Bnew,h_B,colsize,rowsize);
		
	}
    hipMemcpy(Rnew,h_R,sizef,hipMemcpyDeviceToHost);
    hipMemcpy(Gnew,h_G,sizef,hipMemcpyDeviceToHost);
    hipMemcpy(Bnew,h_B,sizef,hipMemcpyDeviceToHost);
    hipFree(h_Rnew); hipFree(h_R);
    hipFree(h_Gnew); hipFree(h_G);
    hipFree(h_Bnew); hipFree(h_B);

	gettimeofday(&tim, NULL);
	double t2=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf("%.6lf seconds elapsed\n", t2-t1);
}

