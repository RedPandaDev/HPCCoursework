#include <stdio.h>
#include <string.h>
#include <sys/types.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ 
void doBlur(int *R,int *h_R,int *G,int *h_G,int *B,int *h_B,int colsize,int rowsize){
	int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;


		if(col<colsize && row<rowsize){
				if (row != 0 && row != (rowsize-1) && col != 0 && col != (colsize-1)){

					R[row * colsize + col] = (h_R[(row+1) * colsize + col]+h_R[(row-1) * colsize + col]+h_R[row * colsize + (col+1)]+h_R[row * colsize + (col-1)])/4;
					G[row * colsize + col] = (h_G[(row+1) * colsize + col]+h_G[(row-1) * colsize + col]+h_G[row * colsize + (col+1)]+h_G[row * colsize + (col-1)])/4;
					B[row * colsize + col] = (h_B[(row+1) * colsize + col]+h_B[(row-1) * colsize + col]+h_B[row * colsize + (col+1)]+h_B[row * colsize + (col-1)])/4;
					// R[row * colsize + col] = (h_R[(row+1) * colsize + col]+h_R[(row-1) * colsize + col]+h_R[row * colsize + (col+1)]+h_R[row * colsize + (col-1)])*0;
					// G[row * colsize + col] = (h_G[(row+1) * colsize + col]+h_G[(row-1) * colsize + col]+h_G[row * colsize + (col+1)]+h_G[row * colsize + (col-1)])*0;
					// B[row * colsize + col] = (h_B[(row+1) * colsize + col]+h_B[(row-1) * colsize + col]+h_B[row * colsize + (col+1)]+h_B[row * colsize + (col-1)])*0;

				}
				else if (row == 0 && col != 0 && col != (colsize-1)){
					R[row * colsize + col] = (h_R[(row+1) * colsize + col]+h_R[row * colsize + (col+1)]+h_R[row * colsize + (col-1)])/3;
					G[row * colsize + col] = (h_G[(row+1) * colsize + col]+h_G[row * colsize + (col+1)]+h_G[row * colsize + (col-1)])/3;
					B[row * colsize + col] = (h_B[(row+1) * colsize + col]+h_B[row * colsize + (col+1)]+h_B[row * colsize + (col-1)])/3;
					// R[row * colsize + col] = (h_R[(row+1) * colsize + col]+h_R[row * colsize + (col+1)]+h_R[row * colsize + (col-1)])*0;
					// G[row * colsize + col] = (h_G[(row+1) * colsize + col]+h_G[row * colsize + (col+1)]+h_G[row * colsize + (col-1)])*0;
					// B[row * colsize + col] = (h_B[(row+1) * colsize + col]+h_B[row * colsize + (col+1)]+h_B[row * colsize + (col-1)])*0;
					
				}
				else if (row == (rowsize-1) && col != 0 && col != (colsize-1)){
					R[row * colsize + col] = (h_R[(row-1) * colsize + col]+h_R[row * colsize + (col+1)]+h_R[row * colsize + (col-1)])/3;
					G[row * colsize + col] = (h_G[(row-1) * colsize + col]+h_G[row * colsize + (col+1)]+h_G[row * colsize + (col-1)])/3;
					B[row * colsize + col] = (h_B[(row-1) * colsize + col]+h_B[row * colsize + (col+1)]+h_B[row * colsize + (col-1)])/3;
					
				}
				else if (col == 0 && row != 0 && row != (rowsize-1)){
					R[row * colsize + col] = (h_R[(row+1) * colsize + col]+h_R[(row-1) * colsize + col]+h_R[row * colsize + (col+1)])/3;
					G[row * colsize + col] = (h_G[(row+1) * colsize + col]+h_G[(row-1) * colsize + col]+h_G[row * colsize + (col+1)])/3;
					B[row * colsize + col] = (h_B[(row+1) * colsize + col]+h_B[(row-1) * colsize + col]+h_B[row * colsize + (col+1)])/3;
					
				}
				else if (col == (colsize-1) && row != 0 && row != (rowsize-1)){
					R[row * colsize + col] = (h_R[(row+1) * colsize + col]+h_R[(row-1) * colsize + col]+h_R[row * colsize + (col-1)])/3;
					G[row * colsize + col] = (h_G[(row+1) * colsize + col]+h_G[(row-1) * colsize + col]+h_G[row * colsize + (col-1)])/3;
					B[row * colsize + col] = (h_B[(row+1) * colsize + col]+h_B[(row-1) * colsize + col]+h_B[row * colsize + (col-1)])/3;
					
				}
				else if (row==0 &&col==0){
					R[row * colsize + col] = (h_R[row * colsize + (col+1)]+h_R[(row+1) * colsize + col])/2;
					G[row * colsize + col] = (h_G[row * colsize + (col+1)]+h_G[(row+1) * colsize + col])/2;
					B[row * colsize + col] = (h_B[row * colsize + (col+1)]+h_B[(row+1) * colsize + col])/2;
					
				}
				else if (row==0 &&col==(colsize-1)){
					R[row * colsize + col] = (h_R[row * colsize + (col-1)]+h_R[(row+1) * colsize + col])/2;
					G[row * colsize + col] = (h_G[row * colsize + (col-1)]+h_G[(row+1) * colsize + col])/2;
					B[row * colsize + col] = (h_B[row * colsize + (col-1)]+h_B[(row+1) * colsize + col])/2;
					
				}
				else if (row==(rowsize-1) &&col==0){
					R[row * colsize + col] = (h_R[row * colsize + (col+1)]+h_R[(row-1) * colsize + col])/2;
					G[row * colsize + col] = (h_G[row * colsize + (col+1)]+h_G[(row-1) * colsize + col])/2;
					B[row * colsize + col] = (h_B[row * colsize + (col+1)]+h_B[(row-1) * colsize + col])/2;
					
				}
				else if (row==(rowsize-1) &&col==(colsize-1)){
					R[row * colsize + col] = (h_R[row * colsize + (col-1)]+h_R[(row-1) * colsize + col])/2;
					G[row * colsize + col] = (h_G[row * colsize + (col-1)]+h_G[(row-1) * colsize + col])/2;
					B[row * colsize + col] = (h_B[row * colsize + (col-1)]+h_B[(row-1) * colsize + col])/2;
					
				}	




			}
		
}
__global__ 
void doCopy(int *R,int *h_R,int *G,int *h_G,int *B,int *h_B,int colsize,int rowsize){
	int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

	if(col<colsize && row<rowsize){
		h_R[row * colsize + col] = R[row * colsize + col];
		h_G[row * colsize + col] = G[row * colsize + col];
		h_B[row * colsize + col] = B[row * colsize + col];

	}
	

}

int main (int argc, const char * argv[]) {
	static int const maxlen = 200, rowsize = 521, colsize = 428, linelen = 12;
	char str[maxlen], lines[5][maxlen];
	FILE *fp, *fout;
	int nlines = 0;
	unsigned int h1, h2, h3;
	char *sptr;
	int row = 0, col = 0, nblurs, lineno=0, k;
	struct timeval tim;
	int *R, *B, *G;
   	int sizei; 
   	sizei = sizeof(int)*colsize*rowsize;

   	R = (int*)malloc(sizei);
   	G = (int*)malloc(sizei);
   	B = (int*)malloc(sizei);
	
	fp = fopen("David.ps", "r");
 
	while(! feof(fp))
	{
		fscanf(fp, "\n%[^\n]", str);
		if (nlines < 5) {strcpy((char *)lines[nlines++],(char *)str);}

		else{
			for (sptr=&str[0];*sptr != '\0';sptr+=6){
				sscanf(sptr,"%2x",&h1);
				sscanf(sptr+2,"%2x",&h2);
				sscanf(sptr+4,"%2x",&h3);

				if (col==colsize){
					col = 0;
					row++;
				}
				if (row < rowsize && col < colsize) {

					R[row * colsize + col] = h1;

					G[row * colsize + col] = h2;

					B[row * colsize + col] = h3;
				}
				col++;
			}
		}

	}
	fclose(fp);
	
	nblurs = 160;
	gettimeofday(&tim, NULL);
	double t1=tim.tv_sec+(tim.tv_usec/1000000.0);

	int *Rnew, *Bnew, *Gnew;
	int *h_R, *h_G, *h_B;

	h_R = (int*)malloc(sizei);

	h_G = (int*)malloc(sizei);
		
	h_B = (int*)malloc(sizei);

	// memset(h_R, 0, sizeof h_R);
	// memset(R, 0, sizeof R);

	// memset(h_G, 0, sizeof h_G);
	// memset(G, 0, sizeof G);

	// memset(h_B, 0, sizeof h_B);
	// memset(B, 0, sizeof B);


	Rnew = (int*)malloc(sizei);
   	Gnew = (int*)malloc(sizei);
   	Bnew = (int*)malloc(sizei);

   	int *d_R, *d_G, *d_B;


	hipMalloc((void **)&h_R,sizei);
	hipMalloc((void **)&h_G,sizei);
	hipMalloc((void **)&h_B,sizei);

	hipMalloc((void **)&d_R,sizei);
	hipMalloc((void **)&d_G,sizei);
	hipMalloc((void **)&d_B,sizei);

	hipMemcpy(h_R,R,sizei,hipMemcpyHostToDevice);
	hipMemcpy(h_G,G,sizei,hipMemcpyHostToDevice);
	hipMemcpy(h_B,B,sizei,hipMemcpyHostToDevice);


	dim3 dimGrid(ceil(colsize/(float)32),ceil(rowsize/(float)32),1);
    dim3 dimBlock(32,32,1);


	for(k=0;k<nblurs;k++){

		doBlur<<<dimGrid,dimBlock>>>(d_R,h_R,d_G,h_G,d_B,h_B,colsize,rowsize);
        doCopy<<<dimGrid,dimBlock>>>(d_R,h_R,d_G,h_G,d_B,h_B,colsize,rowsize);	
	}



	hipMemcpy(Rnew,h_R,sizei,hipMemcpyDeviceToHost);
	hipMemcpy(Gnew,h_G,sizei,hipMemcpyDeviceToHost);
	hipMemcpy(Bnew,h_B,sizei,hipMemcpyDeviceToHost);

	hipFree(h_R);
	hipFree(h_G);
	hipFree(h_B);

	hipFree(d_R);
	hipFree(d_G);
	hipFree(d_B);

	gettimeofday(&tim, NULL);
	double t2=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf("%.6lf seconds elapsed\n", t2-t1);


	fout= fopen("DavidBlur.ps", "w");
	for (k=0;k<nlines;k++) fprintf(fout,"\n%s", lines[k]);
	fprintf(fout,"\n");
	for(row=0;row<rowsize;row++){
		for (col=0;col<colsize;col++){
			fprintf(fout,"%02x%02x%02x",Rnew[row*colsize+col],Gnew[row*colsize+col],Bnew[row*colsize+col]);
			lineno++;
			if (lineno==linelen){
				fprintf(fout,"\n");
				lineno = 0;
			}
		}
	}
	fclose(fout);
    return 0;
}
