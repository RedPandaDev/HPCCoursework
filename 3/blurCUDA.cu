#include <stdio.h>
#include <string.h>
#include <sys/types.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ 
void doBlur(int *h_Rnew,int *h_R,int *h_Gnew,int *h_G,int *h_Bnew,int *h_B,int colsize,int rowsize){
	int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;


		if(col<colsize && row<rowsize){
				if (row != 0 && row != (rowsize-1) && col != 0 && col != (colsize-1)){

					h_Rnew[row * colsize + col] = (h_R[(row+1) * colsize + col]+h_R[(row-1) * colsize + col]+h_R[row * colsize + (col+1)]+h_R[row * colsize + (col-1)])/4;
					h_Gnew[row * colsize + col] = (h_G[(row+1) * colsize + col]+h_G[(row-1) * colsize + col]+h_G[row * colsize + (col+1)]+h_G[row * colsize + (col-1)])/4;
					h_Bnew[row * colsize + col] = (h_B[(row+1) * colsize + col]+h_B[(row-1) * colsize + col]+h_B[row * colsize + (col+1)]+h_B[row * colsize + (col-1)])/4;

				}
				else if (row == 0 && col != 0 && col != (colsize-1)){
					h_Rnew[row * colsize + col] = (h_R[(row+1) * colsize + col]+h_R[row * colsize + (col+1)]+h_R[row * colsize + (col-1)])/3;
					h_Gnew[row * colsize + col] = (h_G[(row+1) * colsize + col]+h_G[row * colsize + (col+1)]+h_G[row * colsize + (col-1)])/3;
					h_Bnew[row * colsize + col] = (h_B[(row+1) * colsize + col]+h_B[row * colsize + (col+1)]+h_B[row * colsize + (col-1)])/3;
					
				}
				else if (row == (rowsize-1) && col != 0 && col != (colsize-1)){
					h_Rnew[row * colsize + col] = (h_R[(row-1) * colsize + col]+h_R[row * colsize + (col+1)]+h_R[row * colsize + (col-1)])/3;
					h_Gnew[row * colsize + col] = (h_G[(row-1) * colsize + col]+h_G[row * colsize + (col+1)]+h_G[row * colsize + (col-1)])/3;
					h_Bnew[row * colsize + col] = (h_B[(row-1) * colsize + col]+h_B[row * colsize + (col+1)]+h_B[row * colsize + (col-1)])/3;
					
				}
				else if (col == 0 && row != 0 && row != (rowsize-1)){
					h_Rnew[row * colsize + col] = (h_R[(row+1) * colsize + col]+h_R[(row-1) * colsize + col]+h_R[row * colsize + (col+1)])/3;
					h_Gnew[row * colsize + col] = (h_G[(row+1) * colsize + col]+h_G[(row-1) * colsize + col]+h_G[row * colsize + (col+1)])/3;
					h_Bnew[row * colsize + col] = (h_B[(row+1) * colsize + col]+h_B[(row-1) * colsize + col]+h_B[row * colsize + (col+1)])/3;
					
				}
				else if (col == (colsize-1) && row != 0 && row != (rowsize-1)){
					h_Rnew[row * colsize + col] = (h_R[(row+1) * colsize + col]+h_R[(row-1) * colsize + col]+h_R[row * colsize + (col-1)])/3;
					
				}
				else if (row==0 &&col==0){
					h_Rnew[row * colsize + col] = (h_R[row * colsize + (col+1)]+h_R[(row+1) * colsize + col])/2;
					
				}
				else if (row==0 &&col==(colsize-1)){
					h_Rnew[row * colsize + col] = (h_R[row * colsize + (col-1)]+h_R[(row+1) * colsize + col])/2;
					
				}
				else if (row==(rowsize-1) &&col==0){
					h_Rnew[row * colsize + col] = (h_R[row * colsize + (col+1)]+h_R[(row-1) * colsize + col])/2;
					
				}
				else if (row==(rowsize-1) &&col==(colsize-1)){
					h_Rnew[row * colsize + col] = (h_R[row * colsize + (col-1)]+h_R[(row-1) * colsize + col])/2;
					
				}		


			}
		
}
__global__ 
void doCopy(int *h_Rnew,int *h_R,int *h_Gnew,int *h_G,int *h_Bnew,int *h_B,int colsize,int rowsize){
	int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;
	if(col<colsize && row<rowsize){
		h_R[row * colsize + col] = h_Rnew[row * colsize + col];
		h_G[row * colsize + col] = h_Gnew[row * colsize + col];
		h_B[row * colsize + col] = h_Bnew[row * colsize + col];

	}
	

}

int main (int argc, const char * argv[]) {
	static int const maxlen = 200, rowsize = 521, colsize = 428, linelen = 12;
	char str[maxlen], lines[5][maxlen];
	FILE *fp, *fout;
	int nlines = 0;
	unsigned int h1, h2, h3;
	char *sptr;
	int row = 0, col = 0, nblurs, lineno=0, k;
	struct timeval tim;
	int *h_R, *h_G, *h_B, *R, *B, *G;
	int *h_Rnew,*h_Gnew,*h_Bnew;
   	int sizei; 
   	sizei = sizeof(int)*colsize*rowsize;

   	R = (int*)malloc(sizei);
   	G = (int*)malloc(sizei);
   	B = (int*)malloc(sizei);

	h_R = (int*)malloc(sizei);
	h_Rnew = (int*)malloc(sizei);

	h_G = (int*)malloc(sizei);
	h_Gnew = (int*)malloc(sizei);
		
	h_B = (int*)malloc(sizei);
	h_Bnew = (int*)malloc(sizei);

	memset(h_R, 0, sizeof h_R);
	memset(h_Rnew, 0, sizeof h_Rnew);

	memset(h_G, 0, sizeof h_G);
	memset(h_Gnew, 0, sizeof h_Gnew);

	memset(h_B, 0, sizeof h_B);
	memset(h_Bnew, 0, sizeof h_Bnew);

	
	fp = fopen("DavidBlur.ps", "r");
 
	while(! feof(fp))
	{
		fscanf(fp, "\n%[^\n]", str);
		if (nlines < 5) {strcpy((char *)lines[nlines++],(char *)str);}

		else{
			for (sptr=&str[0];*sptr != '\0';sptr+=6){
				sscanf(sptr,"%2x",&h1);
				sscanf(sptr+2,"%2x",&h2);
				sscanf(sptr+4,"%2x",&h3);

				if (col==colsize){
					col = 0;
					row++;
				}
				if (row < rowsize && col < colsize) {

					R[row * colsize + col] = h1;

					G[row * colsize + col] = h2;

					B[row * colsize + col] = h3;
				}
				col++;
			}
		}

	}
	fclose(fp);
	
	nblurs = 10;
	gettimeofday(&tim, NULL);
	double t1=tim.tv_sec+(tim.tv_usec/1000000.0);


	hipMalloc((void **)&h_Rnew,sizei);
	hipMalloc((void **)&h_R,sizei);

	hipMalloc((void **)&h_Gnew,sizei);
	hipMalloc((void **)&h_G,sizei);

	hipMalloc((void **)&h_Bnew,sizei);
	hipMalloc((void **)&h_B,sizei);

	// cudaMemcpy(h_R,R,sizei,cudaMemcpyHostToDevice);
	// cudaMemcpy(h_G,G,sizei,cudaMemcpyHostToDevice);
	// cudaMemcpy(h_B,B,sizei,cudaMemcpyHostToDevice);


	dim3 dimGrid(ceil(colsize/(int)16),ceil(rowsize/(int)32),1);
    dim3 dimBlock(16,32,1);


	for(k=0;k<nblurs;k++){

		doBlur<<<dimGrid,dimBlock>>>(R,h_R,G,h_G,B,h_B,colsize,rowsize);
        doCopy<<<dimGrid,dimBlock>>>(R,h_R,G,h_G,B,h_B,colsize,rowsize);		
	}

	// cudaMemcpy(R,h_R,sizei,cudaMemcpyHostToDevice);
	// cudaMemcpy(G,h_G,sizei,cudaMemcpyHostToDevice);
	// cudaMemcpy(B,h_B,sizei,cudaMemcpyHostToDevice);


	hipFree(h_Rnew); hipFree(h_R);
	hipFree(h_Gnew); hipFree(h_G);
	hipFree(h_Bnew); hipFree(h_B);

	gettimeofday(&tim, NULL);
	double t2=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf("%.6lf seconds elapsed\n", t2-t1);


	fout= fopen("DavidBlur.ps", "w");
	for (k=0;k<nlines;k++) fprintf(fout,"\n%s", lines[k]);
	fprintf(fout,"\n");
	for(row=0;row<rowsize;row++){
		for (col=0;col<colsize;col++){
			fprintf(fout,"%02x%02x%02x",R[row*colsize+col],G[row*colsize+col],B[row*colsize+col]);
			lineno++;
			if (lineno==linelen){
				fprintf(fout,"\n");
				lineno = 0;
			}
		}
	}
	fclose(fout);
    return 0;
}
