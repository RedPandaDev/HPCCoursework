#include <stdio.h>
#include <string.h>
#include <sys/types.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <math.h>

void doBlur(int h_Rnew,int h_R,int colsize,int rowsize){
	int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

		if(col<colsize && row<rowsize){
				if (row != 0 && row != (rowsize-1) && col != 0 && col != (colsize-1)){
					h_Rnew[row * colsize + col] = (h_R[(row+1) * colsize + col]+h_R[(row-1) * colsize + col]+h_R[row * colsize + (col+1)]+h_R[row * colsize + (col-1)])/4;
				}
				else if (row == 0 && col != 0 && col != (colsize-1)){
					h_Rnew[row * colsize + col] = (h_R[(row+1) * colsize + col]+h_R[row * colsize + (col+1)]+h_R[row * colsize + (col-1)])/3;
					
				}
				else if (row == (rowsize-1) && col != 0 && col != (colsize-1)){
					h_Rnew[row * colsize + col] = (h_R[(row-1) * colsize + col]+h_R[row * colsize + (col+1)]+h_R[row * colsize + (col-1)])/3;
					
				}
				else if (col == 0 && row != 0 && row != (rowsize-1)){
					h_Rnew[row * colsize + col] = (h_R[(row+1) * colsize + col]+h_R[(row-1) * colsize + col]+h_R[row * colsize + (col+1)])/3;
					
				}
				else if (col == (colsize-1) && row != 0 && row != (rowsize-1)){
					h_Rnew[row * colsize + col] = (h_R[(row+1) * colsize + col]+h_R[(row-1) * colsize + col]+h_R[row * colsize + (col-1)])/3;
					
				}
				else if (row==0 &&col==0){
					h_Rnew[row * colsize + col] = (h_R[row * colsize + (col+1)]+h_R[(row+1) * colsize + col])/2;
					
				}
				else if (row==0 &&col==(colsize-1)){
					h_Rnew[row * colsize + col] = (h_R[row * colsize + (col-1)]+h_R[(row+1) * colsize + col])/2;
					
				}
				else if (row==(rowsize-1) &&col==0){
					h_Rnew[row * colsize + col] = (h_R[row * colsize + (col+1)]+h_R[(row-1) * colsize + col])/2;
					
				}
				else if (row==(rowsize-1) &&col==(colsize-1)){
					h_Rnew[row * colsize + col] = (h_R[row * colsize + (col-1)]+h_R[(row-1) * colsize + col])/2;
					
				}		
			}
		
}

void doCopy(int h_Rnew,int h_R,int colsize,int rowsize){
	int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;
	if(col<colsize && row<rowsize){
	    h_R[row * colsize + col] = h_Rnew[row * colsize + col];

	}

	}

int RGBval(int x){

	int r,g,b, pow8 = 256;
    if(x<=0.5){
        b = (int)((1.0-2.0*x)*255.0);
        g = (int)(2.0*x*255.0);
		r = 0; 
    }
    else{
        b = 0;
        g = (int)((2.0-2.0*x)*255.0);
        r = (int)((2.0*x-1.0)*255.0);
    }
    return (b+(g+r*pow8)*pow8);
}

int main (int argc, const char * argv[]) {
	static int const maxlen = 200, rowsize = 521, colsize = 428, linelen = 12;
	char str[maxlen], lines[5][maxlen];
	FILE *fp, *fout;
	int nlines = 0;
	unsigned int h1, h2, h3;
	char *sptr;
	int *R, *G, *B;
	int row = 0, col = 0, nblurs, lineno=0, k;
	struct timeval tim;
	
	fp = fopen("David.ps", "r");
 
	while(! feof(fp))
	{
		fscanf(fp, "\n%[^\n]", str);
		if (nlines < 5) {strcpy((char *)lines[nlines++],(char *)str);}
		else{
			for (sptr=&str[0];*sptr != '\0';sptr+=6){
				sscanf(sptr,"%2x",&h1);
				sscanf(sptr+2,"%2x",&h2);
				sscanf(sptr+4,"%2x",&h3);
				
				if (col==colsize){
					col = 0;
					row++;
				}
				if (row < rowsize && col < colsize) {
					R[row * colsize + col] = h1;
					G[row * colsize + col] = h2;
					B[row * colsize + col] = h3;
				}
				col++;
			}
		}
	}
	fclose(fp);
	
	nblurs = 10;
	gettimeofday(&tim, NULL);
	double t1=tim.tv_sec+(tim.tv_usec/1000000.0);
	int *h_R;
   	int *h_Rnew;
   	int sizei; 
   	sizei = sizeof(int)*colsize*rowsize;

	h_R = (int*)malloc(sizei);
	h_Rnew = (int*)malloc(sizei);

	memset(h_R, 0, sizeof h_R);

	hipMalloc((void **)&h_Rnew,sizei);
	hipMalloc((void **)&h_R,sizei);
	hipMemcpy(h_R,R,sizei,hipMemcpyHostToDevice);
   	
	for(k=0;k<nblurs;k++){

		doBlur(h_Rnew,h_R,colsize,rowsize);
        doCopy(h_Rnew,h_R,colsize,rowsize);
		
	}


    hipMemcpy(R,h_R,sizei,hipMemcpyDeviceToHost);
	hipFree(h_Rnew); hipFree(h_R);


	gettimeofday(&tim, NULL);
	double t2=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf("%.6lf seconds elapsed\n", t2-t1);
	
	fout= fopen("DavidBlur.ps", "w");
	for (k=0;k<nlines;k++) fprintf(fout,"\n%s", lines[k]);
	fprintf(fout,"\n");
	for(row=0;row<rowsize;row++){
		for (col=0;col<colsize;col++){
			fprintf(fout,"%06x",RGBval(h_R[row*colsize+col]));
			lineno++;
			if (lineno==linelen){
				fprintf(fout,"\n");
				lineno = 0;
			}
		}
	}
	fclose(fout);
    return 0;
}
